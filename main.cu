#include "hip/hip_runtime.h"
#include <vector>
#include <iostream>
#include <numeric>
#include <iostream>
#include <fstream>
#include <string>
#include <cmath>
#include <algorithm>
#include <hip/hip_runtime.h>
using namespace std;

// void printMatrix(const vector<vector<float> >& matrix) {
//     for (const auto& row : matrix) {
//         for (const auto& elem : row) {
//             cout << elem << " ";
//         }
//         cout << endl;
//     }
// }
__global__ void conv(const float* input, const float* kernel, float* output, int isize, int ksize) {
    int i = blockIdx.x * blockDim.x + threadIdx.x; // Column index
    int j = blockIdx.y * blockDim.y + threadIdx.y; // Row index
    int res = isize - ksize + 1;

    if (i < res && j < res) {
        float sum = 0.0f;
        for (int m = 0; m < ksize; ++m) {
            for (int n = 0; n < ksize; ++n) {
                int X = i + m;
                int Y = j + n;
                sum += input[Y * isize + X] * kernel[m * ksize + n];
            }
        }
        output[j * res + i] = sum;
    }
}


// vector<vector<float> > convpad(vector<vector<float> > input, vector<vector<float> > kernel){
//     int isize = input.size();
//     int ksize = kernel.size();
//     int pad = (ksize-1)/2;
//     int padinput = isize + pad*2;
//     int res = isize;

//     vector<vector<float> > padMatrix(padinput, vector<float>(padinput, 0));

//     vector<vector<float> > outputMatrix(res, vector<float>(res, 0));

//     for (int i = 0; i < isize; ++i) {
//         for (int j = 0; j < isize; ++j) {
//             padMatrix[i + pad][j + pad] = input[i][j];
//         }
//     }
//     printMatrix(padMatrix);

//     for (int i = 0; i < res; ++i) {
//         for (int j = 0; j < res; ++j) {
//             float sum = 0;
//             for (int m = 0; m < ksize; ++m) {
//                 for (int n = 0; n < ksize; ++n) {
//                     sum += padMatrix[i + m][j + n] * kernel[m][n];
//                 }
//             }
//             outputMatrix[i][j] = sum;
//         }
//     }

//     return outputMatrix;
// }

__global__ void relu(float* input, float* output, int dataSize) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < dataSize) {
        output[idx] = max(0.0f, input[idx]);
    }
}

__global__ void tanh(float* input, int dataSize) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < dataSize) {
        input[idx] = tanhf(input[idx]); // Note: Use tanhf for float
    }
}


// __global__ void maxPool(const float* input, float* output, int isize, int poolSize, int outputSize) {
//     int ox = blockIdx.x * blockDim.x + threadIdx.x; // Output x-coordinate
//     int oy = blockIdx.y * blockDim.y + threadIdx.y; // Output y-coordinate

//     if (ox < outputSize && oy < outputSize) {
//         float maxVal = -FLT_MAX;
//         for (int i = 0; i < poolSize; ++i) {
//             for (int j = 0; j < poolSize; ++j) {
//                 int ix = ox * poolSize + i;
//                 int iy = oy * poolSize + j;
//                 maxVal = max(maxVal, input[iy * isize + ix]);
//             }
//         }
//         output[oy * outputSize + ox] = maxVal;
//     }
// }


float sigmoid(float x){
    return 1.0f/(1.0f+exp(-x));
}

vector<float> sigfunc(const vector<float>& input){
    vector<float> output(input.size());
    transform(input.begin(), input.end(), output.begin(), sigmoid);
    return output;
}


// vector<float> softmax(const vector<float>& input) {
//     vector<float> outputMatrix(input.size());
//     float p = *max_element(input.begin(), input.end());
//     float sum = 0.0f;
//     for (int i = 0; i < input.size(); ++i) {
//         outputMatrix[i] = exp(input[i] - p); 
//         sum += outputMatrix[i];
//     }
//     for (float& value : outputMatrix) {
//         value /= sum;
//     }

//     return outputMatrix;
// }
vector<vector<float> > fileread(ifstream& file) {
    int rows, cols;
    file >> rows >> cols; 
    vector<vector<float> > matrix(rows, vector<float>(cols));
    for (int i = 0; i < rows; ++i) {
        for (int j = 0; j < cols; ++j) {
            file >> matrix[i][j];
        }
    }
    return matrix;
}

int main() {
    ifstream file("matrix.txt");
    if (!file.is_open()) {
        std::cerr << "Error opening file" << std::endl;
        return -1;
    }
    
    vector<vector<float> > input = fileread(file);
    vector<vector<float> > kernel = fileread(file);
    int isize = input.size();
    int ksize = kernel.size();
    
    vector<vector<float> > outputMatrix(res, vector<float>(res, 0));
    int res = isize - ksize +1;
    size_t inputSize = isize * isize * sizeof(float);
    size_t kernelsize = ksize*ksize*sizeof(float);
    size_t outputsize = res*res*sizeof(float);

    
    float *c_input, *c_kernel, *c_output;
    hipMalloc(&c_input, inputSize);
    hipMalloc(&c_kernel, kernelsize);
    hipMalloc(&c_output, outputsize);


    hipMemcpy(c_input, input, inputsize, hipMemcpyHostToDevice);
    hipMemcpy(c_kernel, kernel, kernelsize, hipMemcpyHostToDevice);

    dim3 threads(16, 16);
    dim3 blocks((res + threads.x - 1) / threads.x, 
                   (res + threads.y - 1) / threads.y);

    conv<<<blocks, threads>>>(c_input, c_kernel, c_output, isize, ksize);

    hipMemcpy(outputMatrix, c_output, outputsize, hipMemcpyDeviceToHost);
    hipFree(c_input);
    hipFree(c_kernel);
    hipFree(c_output);

    printMatrix(outputMatrix);

    return 0;
}

